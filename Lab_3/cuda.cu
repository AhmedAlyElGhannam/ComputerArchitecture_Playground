#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <fstream>

using namespace std;

// try N = 5, 50, 500
#define N 2

// available matrix operations
typedef enum
{
    ADD = 1,
    MUL = 2
} matOp;

__global__ void matrixMulCUDA(int* matrixA, int* matrixB, int* matrixRes) 
{
    // Compute each thread's global row and column index
    int rowIndex = blockIdx.y * blockDim.y + threadIdx.y;
    int colIndex = blockIdx.x * blockDim.x + threadIdx.x;

    // Iterate over row, and down column
    matrixRes[rowIndex][colIndex] = 0;
    for (int k = 0; k < N; k++) 
    {
        // Accumulate results for a single element
        matrixRes[rowIndex][colIndex] += matrixA[rowIndex][k] * matrixB[k][colIndex];
    }
}
/**
if condition to see if row && col are less than N
then simply add
*/
__global__ void matrixAddCUDA(int* matrixA, int* matrixB, int* matrixRes) 
{
    // Compute each thread's global row and column index
    int rowIndex = blockIdx.y * blockDim.y + threadIdx.y;
    int colIndex = blockIdx.x * blockDim.x + threadIdx.x;

    // simply add
    matrixRes[rowIndex * N + colIndex] = matrixA[rowIndex * N + colIndex] + matrixB[rowIndex * N + colIndex];
}

// h_sth => host variable (PC)
// d_sth => device variable (GPU)

void matrixOperationCudaWrapper(const int (&h_matrixA)[N][N], const int (&h_matrixB)[N][N], int (&h_matrixRes)[N][N], unsigned char operation)
{
    // create pointers to gpu
    int* d_cudaA = 0;
    int* d_cudaB = 0;
    int* d_cudaRes = 0;

    // defining size
    size_t sizeInBytes = N * N * sizeof(int);

    // allocate memory in gpu
    hipMalloc((void**)(&d_cudaA), sizeInBytes);
    hipMalloc((void**)(&d_cudaB), sizeInBytes);
    hipMalloc((void**)(&d_cudaRes), sizeInBytes);

    // copy vectors into gpu hipMemcpy(d_input, inputImg.data, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_cudaA, &h_matrixA, sizeInBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_cudaB, &h_matrixB, sizeInBytes, hipMemcpyHostToDevice);

    // defining CTA and grid dimensions
    int threads = 16;
    int blocks = (N + threads - 1) / threads;

    // setting up kernel launch parameters
    dim3 BLOCKS(blocks, blocks);
    dim3 THREADS(threads, threads);

    // launch kernel for chosen operation
    if (operation == ADD)
        matrixAddCUDA<<<BLOCKS, THREADS>>>(d_cudaA, d_cudaB, d_cudaRes);
    else if (operation == MUL)
        matrixMulCUDA<<<BLOCKS, THREADS>>>(d_cudaA, d_cudaB, d_cudaRes);
    else
    {
        cout << "yo, what the duck?" << endl;
        return; // do not continue this mess!
    }
        
    // copy result from gpu memory
    hipMemcpy(&h_matrixRes, d_cudaRes, sizeInBytes, hipMemcpyDeviceToHost);

    // free allocated gpu memory
    hipFree(d_cudaA);
    hipFree(d_cudaB);
    hipFree(d_cudaRes);

    return;
}

void populateMatrix(int (&matrix)[N][N]) 
{
    for (int i = 0; i < N; i++) 
    {
        for (int j = 0; j < N; j++)
        {
            matrix[i][j] = rand() % 100; // Generate random numbers between 0 and N-1
        }
    }
}

void printMatrix(const int (&matrix)[N][N]) 
{
    for (int i = 0; i < N; i++) 
    {
        for (int j = 0; j < N; j++)
        {
            cout << matrix[i][j] << "\t";
        }
        cout << endl;
    }
}

void csvMatrix(const int (&matrix)[N][N], const char *filename) {
    std::ofstream file(filename);
    if (!file.is_open()) 
    {
        std::cerr << "Error opening file " << filename << std::endl;
        return;
    }

    for (int i = 0; i < N; i++) 
    {
        for (int j = 0; j < N; j++) 
        {
            file << matrix[i][j];
            if (j < N - 1) 
            {
                file << ",";
            }
        }
        file << "\n";
    }
    file.close();
}

int main() 
{
    // random number generation shenanigans
    srand(time(NULL));

    // define matrices
    int matA[N][N];
    int matB[N][N];
    int matC[N][N];
    int matRes[N][N];
    int matTemp[N][N];

    // populate matrix A && B
    populateMatrix(matA);
    populateMatrix(matB);
    populateMatrix(matC);

    // output matrix A && B as csv files for references
    csvMatrix(matA, "MatrixA.csv");
    csvMatrix(matB, "MatrixB.csv");
    csvMatrix(matC, "MatrixC.csv");
    
    // // C * ((A * B) + (B * A))
    // multMatrix(matA, matB, matRes); // A * B = res
    
    // multMatrix(matB, matA, matTemp); // B * A = temp
    
    // addMatrix(matRes, matTemp, matTemp); // res + temp = temp

    // multMatrix(matC, matTemp, matRes); // C * temp = res
    
    // // output matrix C result for reference
    // csvMatrix(matRes, "Result.csv");

    return 0;
}